/**
 * Lecture 3 - Euclidean Vector Norm
 *
 * Implement a CUDA code that calculates the sumatory of the square for several
 * vectors on the GPU, which can be later used to calculate the euclidean norm
 * (if applying the square root)
 *
 * As input only one matrix of size "mxn" is created (floats). It represents
 * “m” vectors (rows) of length “n”
 *
 * As output you should provide a vector with “n” floats (element “i” is the
 * summatory of vector “i”)
 *
 * Each thread calculates one output element (the whole calculation for one vector)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <sys/resource.h>

// Values for MxN matrix
const int DEFAULT_M = 100;
const int DEFAULT_N = 200;

const int DEFAULT_BLOCK_SIZE = 128; // Default CUDA block size
const float ZERO_VAL = 0.0f;

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                                     \
    do                                                                                           \
    {                                                                                            \
        hipError_t cuErr = call;                                                                \
        if (hipSuccess != cuErr)                                                                \
        {                                                                                        \
            printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr)); \
            exit(0);                                                                             \
        }                                                                                        \
    } while (0)

/**
 * Para medir el tiempo transcurrido (elapsed time):
 *
 * resnfo: tipo de dato definido para abstraer la métrica de recursos a usar
 * timenfo: tipo de dato definido para abstraer la métrica de tiempo a usar
 *
 * timestamp: abstrae función usada para tomar las muestras del tiempo transcurrido
 *
 * printtime: abstrae función usada para imprimir el tiempo transcurrido
 *
 * void myElapsedtime(resnfo start, resnfo end, timenfo *t): función para obtener
 * el tiempo transcurrido entre dos medidas
 */
#ifdef _noWALL_
typedef struct rusage resnfo;
typedef struct _timenfo
{
    double time;
    double systime;
} timenfo;
#define timestamp(sample) getrusage(RUSAGE_SELF, (sample))
#define printtime(t) printf("%15f s (%f user + %f sys) ", \
                            t.time + t.systime, t.time, t.systime);
#else
typedef struct timeval resnfo;
typedef double timenfo;
#define timestamp(sample) gettimeofday((sample), 0)
#define printtime(t) printf("%15f s ", t);
#endif

void myElapsedtime(const resnfo start, const resnfo end, timenfo *const t)
{
#ifdef _noWALL_
    t->time = (end.ru_utime.tv_sec + (end.ru_utime.tv_usec * 1E-6)) - (start.ru_utime.tv_sec + (start.ru_utime.tv_usec * 1E-6));
    t->systime = (end.ru_stime.tv_sec + (end.ru_stime.tv_usec * 1E-6)) - (start.ru_stime.tv_sec + (start.ru_stime.tv_usec * 1E-6));
#else
    *t = (end.tv_sec + (end.tv_usec * 1E-6)) - (start.tv_sec + (start.tv_usec * 1E-6));
#endif /*_noWALL_*/
}

/**
 * Populate the matrix with values for the tests
 */
void populate_matrix(float *matrix, const unsigned int m, const unsigned int n)
{
    unsigned int i, j;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            matrix[i * n + j] = ((i + 1) + j) % 10;
        }
    }
}

/**
 * Prints the values of the matrix to the screen
 */
void print_matrix(float *matrix, const unsigned int m, const unsigned int n)
{
    unsigned int i, j;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", matrix[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

/**
 * Populate the array with the value
 */
void populate_array(float *array, const unsigned int m, const float value)
{
    unsigned int i;
    for (i = 0; i < m; i++)
    {
        array[i] = value;
    }
}

/**
 * Prints the values of the array to the screen
 */
void print_array(float *array, const unsigned int m)
{
    unsigned int i;
    for (i = 0; i < m; i++)
    {
        printf("%f ", array[i]);
    }
    printf("\n");
}

/**
 * Function that comprate the elements of two matrix's
 */
bool compare_matrix(float *m1, float *m2, const unsigned int m, const unsigned int n)
{
    unsigned int i, j;
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            if (m1[i * n + j] != m2[i * n + j])
                return false;
        }
    }
    return true;
}

/**
 * Function that comprate the elements of two array's
 */
bool compare_array(float *a1, float *a2, const unsigned int m)
{
    for (int i = 0; i < m; i++)
        if (a1[i] != a2[i])
        {
            printf("Mismatch at index %d, was: %f, should be: %f\n", i, a1[i], a2[i]);
            return false;
        }
    return true;
}

// CPU execution
// ============================================================================

/**
 * Eucliden vec norm function that perform the operation in the CPU
 */
void euclidean_vec_norm_CPU(float *a, float *c, const unsigned int m, const unsigned int n)
{
    resnfo start, end;
    timenfo time;

    unsigned int row, colum;

    timestamp(&start); // Start time measurement
    for (row = 0; row < m; row++)
    {
        for (colum = 0; colum < n; colum++)
        {
            c[row] += (a[row * n + colum] * a[row * n + colum]);
        }
    }
    timestamp(&end); // Stop time measurement
    myElapsedtime(start, end, &time);
    printtime(time);
}

// GPU definition
// ============================================================================

/**
 * Kernel definition
 */
__global__ void euclidean_vec_norm_GPU_kernel(float *a, float *c, const unsigned int m, const unsigned int n)
{
    unsigned int colum;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < m)
    {
        for (colum = 0; colum < n; colum++)
        {
            c[row] += (a[row * n + colum] * a[row * n + colum]);
        }
    }
}

/**
 * Eucliden vec norm function that perform the operation in the GPU
 */
void euclidean_vec_norm_GPU(float *a, float *c, const unsigned int m,
                            const unsigned int n, const unsigned int block_size)
{
    resnfo start, end;
    timenfo time;

    float *d_A, *d_C;

    // Number of bytes to allocate for N vector
    size_t vectorNumBytes = m * sizeof(float);
    // Number of bytes to allocate for MxN matrix
    size_t matrixNumBytes = m * n * sizeof(float);

    // Allocate device memory and copy input data over to GPU
    cudaErrorCheck(hipMalloc(&d_A, matrixNumBytes));
    cudaErrorCheck(hipMalloc(&d_C, vectorNumBytes));

    // Copy data from host matrix A to device matrix
    cudaErrorCheck(hipMemcpy(d_A, a, matrixNumBytes, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_C, c, vectorNumBytes, hipMemcpyHostToDevice));

    // Launch kernel
    //  - threads_per_block: number of CUDA threads per grid block
    //	- blocks_in_grid   : number of blocks in grid
    //	(These are c structs with 3 member variables x, y, x)
    dim3 threads_per_block(1,
                           block_size,
                           1); // dim3 variable holds 3 dimensions
    dim3 blocks_in_grid(1,
                        ceil(float(m) / threads_per_block.y),
                        //(m + threads_per_block.y - 1) / threads_per_block.y,
                        1);

    printf(" threads_per_block         = %d\n", block_size);
    // printf(" blocks_in_grid            = %d\n", (m + threads_per_block.y - 1) / threads_per_block.y);
    printf(" blocks_in_grid (ceil)     = %f\n", ceil(float(m) / threads_per_block.y));

    timestamp(&start); // Start time measurement
    euclidean_vec_norm_GPU_kernel<<<blocks_in_grid, threads_per_block>>>(d_A, d_C, m, n);
    hipDeviceSynchronize();
    timestamp(&end); // Stop time measurement
    myElapsedtime(start, end, &time);
    printtime(time);

    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    hipError_t cuErrSync = hipGetLastError();
    if (cuErrSync != hipSuccess)
    {
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync));
        exit(0);
    }

    // Check for errors on the GPU after control is returned to CPU
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess)
    {
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync));
        exit(0);
    }

    // Copy data from device to CPU
    cudaErrorCheck(hipMemcpy(c, d_C, vectorNumBytes, hipMemcpyDeviceToHost));

    // Free CPU and GPU memory
    cudaErrorCheck(hipFree(d_A));
    cudaErrorCheck(hipFree(d_C));
}

// Main program
// ============================================================================
int main(int argc, char *argv[])
{
    float *h_A, *h_C, *l_A, *l_C;

    printf("--------------------------------\n");
    printf(" Euclidean Vector Norm\n");
    printf("--------------------------------\n");

    // Read from args the matrix size
    unsigned int m = (argc > 1) ? atoi(argv[1]) : DEFAULT_M;
    unsigned int n = (argc > 2) ? atoi(argv[2]) : DEFAULT_N;
    unsigned int block_size = (argc > 3) ? atoi(argv[3]) : DEFAULT_BLOCK_SIZE;

    h_A = new float[m * n];
    h_C = new float[m];
    l_A = new float[m * n];
    l_C = new float[m];

    // Initialize host and local matrix A
    populate_matrix(h_A, m, n);
    populate_matrix(l_A, m, n);
    // print_matrix(h_A, m, n);
    // print_matrix(l_A, m, n);

    if (!compare_matrix(h_A, l_A, m, n))
    {
        printf("ERROR: The host matrix and the local matrix are different!!\n");
        return -1;
    }

    // Initialize host and local array C for results
    populate_array(h_C, m, ZERO_VAL);
    populate_array(l_C, m, ZERO_VAL);
    // print_array(h_C, m);
    // print_array(l_C, m);

    // GPU Execution
    euclidean_vec_norm_GPU(h_A, h_C, m, n, block_size);
    printf(" -> Calculate in the GPU (%d vectors, %d elements with %d threads per block)\n", m, n, block_size);

    // CPU execution
    euclidean_vec_norm_CPU(l_A, l_C, m, n);
    printf(" -> Calculate in the CPU (%d vectors, %d elements with %d threads per block)\n", m, n, block_size);

    // print_array(h_C, m);
    // print_array(l_C, m);

    // Verify results
    if (!compare_array(l_C, h_C, m))
    {
        return -1;
    }

    int num_devices;
    hipGetDeviceCount(&num_devices);

    for (int i = 0; i < num_devices; i++)
    {
        hipDeviceProp_t dev_property;
        hipGetDeviceProperties(&dev_property, i);

        printf(" Device                    = %d: %s\n", i, dev_property.name);
    }

    printf(" M                         = %d\n", m);
    printf(" N                         = %d\n", n);
    printf("--------------------------------\n");
    printf(" SUCCESS\n");
    printf("--------------------------------\n");

    return 0;
}