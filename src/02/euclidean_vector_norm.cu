/*
 * Lecture 3 - Euclidean Vector Norm
 *
 * Implement a CUDA code that calculates the sumatory of the square for several
 * vectors on the GPU, which can be later used to calculate the euclidean norm
 * (if applying the square root)
 *
 * As input only one matrix of size "mxn" is created (floats). It represents
 * “m” vectors (rows) of length “n”
 *
 * As output you should provide a vector with “n” floats (element “i” is the
 * summatory of vector “i”)
 *
 * Each thread calculates one output element (the whole calculation for one vector)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// Values for MxN matrix
#define M 100
#define N 200

const int block_size = 128; // CUDA maximum is 1024 *total* threads in block
const float A_val = 1.0f;   // Default value for the all matrix elements
const float C_res = 200.0f;   // Expected valur for the result vector

// Kernel definition
// ============================================================================
__global__ void euclidean_vec_norm(float *a, float *c)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < M)
    {
        for (int i = 0; i < N; i++)
        {
            c[row] += (a[row * N + i] * a[row * N + i]);
        }
        
    }
}

// Main program
// ============================================================================
int main()
{
    // Number of bytes to allocate for N vector
    size_t vectorNumBytes = M * sizeof(float);
    // Number of bytes to allocate for MxN matrix
    size_t matrixNumBytes = M * N * sizeof(float);

    float *h_A, *h_C, *d_A, *d_C;

    h_A = new float[M * N];
    h_C = new float[M];

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, matrixNumBytes);
    hipMalloc(&d_C, vectorNumBytes);

    // Initialize host matrix A
    for (int i = 0; i < M * N; i++)
    {
        h_A[i] = A_val;
    }
    // Initialize host array C
    for (int i = 0; i < M; i++)
    {
        h_C[i] = 0;
    }

    // Copy data from host matrix A to device matrix
    hipMemcpy(d_A, h_A, matrixNumBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, vectorNumBytes, hipMemcpyHostToDevice);

    // Launch kernel
    //  - threads_per_block: number of CUDA threads per grid block
    //	- blocks_in_grid   : number of blocks in grid
    //	(These are c structs with 3 member variables x, y, x)
    dim3 threads_per_block(1,
                           block_size,
                           1); // dim3 variable holds 3 dimensions
    dim3 blocks_in_grid(1,
                        ceil(float(M) / threads_per_block.y),
                        1);
    euclidean_vec_norm<<<blocks_in_grid, threads_per_block>>>(d_A, d_C);

    // Copy data from device to CPU
    hipMemcpy(h_C, d_C, vectorNumBytes, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < M; i++)
        if (h_C[i] != C_res)
        {
            printf("Mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], C_res);
            return -1;
        }

    // Free CPU and GPU memory
    hipFree(d_A);
    hipFree(d_C);

    printf("\n--------------------------------\n");
    printf("__SUCCESS__\n");
    printf("--------------------------------\n");
    printf("M                         = %d\n", M);
    printf("N                         = %d\n", N);
    printf("Threads Per Block (y-dim) = %d\n", threads_per_block.y);
    printf("Blocks In Grid (y-dim)    = %d\n", blocks_in_grid.y);
    printf("--------------------------------\n\n");

    return 0;
}
