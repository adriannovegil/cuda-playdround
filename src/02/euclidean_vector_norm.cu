/*
 * Lecture 3 - Euclidean Vector Norm
 *
 * Implement a CUDA code that calculates the sumatory of the square for several
 * vectors on the GPU, which can be later used to calculate the euclidean norm
 * (if applying the square root)
 *
 * As input only one matrix of size "mxn" is created (floats). It represents
 * “m” vectors (rows) of length “n”
 *
 * As output you should provide a vector with “n” floats (element “i” is the
 * summatory of vector “i”)
 *
 * Each thread calculates one output element (the whole calculation for one vector)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// Values for MxN matrix
const int DEFAULT_M = 100;
const int DEFAULT_N = 200;

const int DEFAULT_BLOCK_SIZE = 128; // Default CUDA block size
const float A_val = 1.0f;   // Default value for the all matrix elements

// Kernel definition
// ============================================================================
__global__ void euclidean_vec_norm_GPU(float *a, float *c, int m, int n)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < m)
    {
        for (int i = 0; i < n; i++)
        {
            c[row] += (a[row * n + i] * a[row * n + i]);
        }
    }
}

// Main program
// ============================================================================
int main(int argc, char *argv[])
{

    // Read from args the matrix size
    unsigned int m = (argc > 1) ? atoi(argv[1]) : DEFAULT_M;
    unsigned int n = (argc > 2) ? atoi(argv[2]) : DEFAULT_N;
    unsigned int block_size = (argc > 3) ? atoi(argv[3]) : DEFAULT_BLOCK_SIZE;

    // Number of bytes to allocate for N vector
    size_t vectorNumBytes = m * sizeof(float);
    // Number of bytes to allocate for MxN matrix
    size_t matrixNumBytes = m * n * sizeof(float);

    float *h_A, *h_C, *d_A, *d_C;

    h_A = new float[m*n];
    h_C = new float[m];

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, matrixNumBytes);
    hipMalloc(&d_C, vectorNumBytes);

    // Initialize host matrix A
    for (int i = 0; i < m*n; i++)
    {
        h_A[i] = A_val;
    }
    // Initialize host array C
    for (int i = 0; i < m; i++)
    {
        h_C[i] = 0;
    }

    // Copy data from host matrix A to device matrix
    hipMemcpy(d_A, h_A, matrixNumBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, vectorNumBytes, hipMemcpyHostToDevice);

    // Launch kernel
    //  - threads_per_block: number of CUDA threads per grid block
    //	- blocks_in_grid   : number of blocks in grid
    //	(These are c structs with 3 member variables x, y, x)
    dim3 threads_per_block(1,
                           block_size,
                           1); // dim3 variable holds 3 dimensions
    dim3 blocks_in_grid(1,
                        ceil(float(m) / threads_per_block.y),
                        1);
    euclidean_vec_norm_GPU<<<blocks_in_grid, threads_per_block>>>(d_A, d_C, m, n);

    // Copy data from device to CPU
    hipMemcpy(h_C, d_C, vectorNumBytes, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < m; i++)
        if (h_C[i] != (float)n)
        {
            printf("Mismatch at index %d, was: %f, should be: %d\n", i, h_C[i], n);
            return -1;
        }

    // Free CPU and GPU memory
    hipFree(d_A);
    hipFree(d_C);

    printf("\n--------------------------------\n");
    printf("__SUCCESS__\n");
    printf("--------------------------------\n");
    printf("M                         = %d\n", m);
    printf("N                         = %d\n", n);
    printf("Threads Per Block (y-dim) = %d\n", threads_per_block.y);
    printf("Blocks In Grid (y-dim)    = %d\n", blocks_in_grid.y);
    printf("--------------------------------\n\n");

    return 0;
}
